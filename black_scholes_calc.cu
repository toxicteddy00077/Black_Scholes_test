#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <vector>
#include <nlohmann/json.hpp>
#include <fstream>
#include <hip/hip_runtime.h>

typedef struct option {
    float stock_p;
    float strike_p;
    float exp_time;
    float rf_rate;
    float vol;
} opt;

__device__ float cnd(float d) {
    const float A1 = 0.31938153;
    const float A2 = -0.356563782;
    const float A3 = 1.781477937;
    const float A4 = -1.821255978;
    const float A5 = 1.330274429;
    const float RSQRT2PI = 0.39894228040143267793994605993438;

    float K = 1.0 / (1.0 + 0.2316419 * fabs(d));
    float cnd = RSQRT2PI * exp(-0.5 * d * d) *
                (K * (A1 + K * (A2 + K * (A3 + K * (A4 + K * A5)))));

    if (d > 0)
        cnd = 1.0 - cnd;

    return cnd;
}

__global__ void calc(opt* options, float* price) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 10) {
        float S = options[idx].stock_p;
        float X = options[idx].strike_p;
        float T = options[idx].exp_time;
        float R = options[idx].rf_rate;
        float V = options[idx].vol;

        float d1 = (log(S / X) + (R + 0.5 * V * V) * T) / (V * sqrt(T));
        float d2 = d1 - V * sqrt(T);

        float cnd_d1 = cnd(d1);
        float cnd_d2 = cnd(d2);

        price[idx] = S * cnd_d1 - X * exp(-R * T) * cnd_d2;
    }
}

int main() {
    std::ifstream infile("out.json");
    if (!infile.is_open()) {
        std::cerr << "Error opening file" << std::endl;
        return 1;
    }

    std::vector<nlohmann::json> data(10);
    try {
        nlohmann::json jsonData;
        infile >> jsonData;

        if (!jsonData.is_array() || jsonData.size() != 10) {
            std::cerr << "Invalid JSON format or size" << std::endl;
            return 1;
        }

        for (int i = 0; i < 10; i++) {
            data[i] = jsonData[i];
            std::cout << "Read option " << i << ": " << data[i] << std::endl;
        }
    } catch (const nlohmann::json::parse_error& e) {
        std::cerr << "Parse error: " << e.what() << std::endl;
        return 1;
    }

    std::vector<float> h_price(10);
    float* d_price;
    std::vector<opt> options(10);
    for (int i = 0; i < 10; i++) {
        options[i] = {data[i]["stock_p"], data[i]["strike_p"], data[i]["exp_time"], data[i]["rf_rate"], data[i]["vol"]};
    }

    opt* d_options;

    hipMalloc((void**)&d_price, 10 * sizeof(float));
    hipMalloc((void**)&d_options, 10 * sizeof(opt));

    hipMemcpy(d_price, h_price.data(), 10 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_options, options.data(), 10 * sizeof(opt), hipMemcpyHostToDevice);

    int blocksize = 1;
    int gridsize = (10 + blocksize - 1) / blocksize;

    calc<<<gridsize, blocksize>>>(d_options, d_price);

    hipMemcpy(h_price.data(), d_price, 10 * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Result vector (price): ";
    for (int i = 0; i < 10; i++) {
        std::cout << h_price[i] << " ";
    }

    hipFree(d_price);
    hipFree(d_options);

    return 0;
}